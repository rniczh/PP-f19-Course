/*************************************************************************
 * DESCRIPTION:
 *   Parallel Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation by using CUDA
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define ceild(n, d) ceil(((double)(n)) / ((double)(d)))

#define THREADS 96

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void update(void);
void printfinal(void);

int nsteps,                  /* number of time steps */
  tpoints,                 /* total points along string */
  rcode;                   /* generic return code */
float values[MAXPOINTS + 2], /* values at time t */
  oldval[MAXPOINTS + 2],   /* values at time (t-dt) */
  newval[MAXPOINTS + 2];   /* values at time (t+dt) */

/**********************************************************************
 *Checks input values from parameters
 *********************************************************************/
void check_param(void) {
  char tchar[20];

  /* check number of points, number of iterations */
  while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
    printf("Enter number of points along vibrating string [%d-%d]: ", MINPOINTS,
           MAXPOINTS);
    scanf("%s", tchar);
    tpoints = atoi(tchar);
    if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
      printf("Invalid. Please enter value between %d and %d\n", MINPOINTS,
             MAXPOINTS);
  }
  while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
    printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
    scanf("%s", tchar);
    nsteps = atoi(tchar);
    if ((nsteps < 1) || (nsteps > MAXSTEPS))
      printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
  }

  printf("Using points = %d, steps = %d\n", tpoints, nsteps);
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
__device__ float do_math(float val, float old) {
  float dtime, c, dx, tau, sqtau;

  dtime = 0.3;
  c = 1.0;
  dx = 1.0;
  tau = (c * dtime / dx);
  sqtau = tau * tau;
  return (2.0 * val) - old + (sqtau * (-2.0) * val);
}

#define GET_INDEX(nblock) (1 + threadIdx.x + blockIdx.x * nblock)

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void update(float *device_values, int tpoints, int nsteps) {
  int i;
  float values1, newval1, oldval1;
  int idx = GET_INDEX(THREADS); /* k */

  if ((idx == 1) || (idx == tpoints)) {
    values1 = 0.0;
  } else {
    /* initialize this point */
    /* Calculate initial values based on sine curve */
    float x, fac, tmp;
    fac = 2.0 * PI;
    tmp = tpoints - 1;

    /* initialize this point */
    /* Calculate initial values based on sine curve */
    x = (float)(idx - 1)/tmp;
    values1 = sin(fac * x);
    oldval1 = values1;

    /* for each step */
    for (i = 1; i <= nsteps; ++i) {
      /* Update each point for this time step  */
      newval1 = do_math(values1, oldval1);
      oldval1 = values1;
      values1 = newval1;
    }
  }

  device_values[idx] = values1;
}
/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal() {
  int i;

  for (i = 1; i <= tpoints; i++) {
    printf("%6.4f ", values[i]);
    if (i % 10 == 0)
      printf("\n");
  }
}

/**********************************************************************
 *Main program
 *********************************************************************/
int main(int argc, char *argv[]) {
  sscanf(argv[1], "%d", &tpoints);
  sscanf(argv[2], "%d", &nsteps);
  check_param();

  /* setup cuda env */
  float *device_values;
  int size  = (1 + tpoints) * sizeof(float);
  int block = ceild(tpoints, THREADS);

  hipMalloc((void **)&device_values, size);

  printf("Initializing points on the line...\n");
  printf("Updating all points for all time steps...\n");

  update<<<block, THREADS>>>(device_values, tpoints, nsteps);

  /* move result back to host */
  hipMemcpy(values, device_values, size, hipMemcpyDeviceToHost);

  printf("Printing final results...\n");
  printfinal();
  printf("\nDone.\n\n");

  return 0;
}
